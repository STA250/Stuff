#include <stdio.h>

// Note: Needs compute capability > 2.0, so compile with:
// nvcc hello_world_01.cu -arch=compute_20 -code=sm_20,compute_20 -o hello_world_01.out
// Other notes: can have trouble when N is large...
// Default buffer is ~8MB
// See hello_world_02.cu for details.

#include <hip/hip_runtime.h>


#define N 20000
#define GRID_D1 20
#define GRID_D2 2
#define BLOCK_D1 512
#define BLOCK_D2 1
#define BLOCK_D3 1

__global__ void hello(void)
{
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    if (idx < N){  
        printf("Hello world! My block index is (%d,%d) [Grid dims=(%d,%d)], 3D-thread index within block=(%d,%d,%d) => thread index=%d\n", blockIdx.x, blockIdx.y, gridDim.x, gridDim.y, threadIdx.x, threadIdx.y, threadIdx.y, idx);
    } else {
        printf("Hello world! My block index is (%d,%d) [Grid dims=(%d,%d)], 3D-thread index within block=(%d,%d,%d) => thread index=%d [### this thread would not be used for N=%d ###]\n", blockIdx.x, blockIdx.y, gridDim.x, gridDim.y, threadIdx.x, threadIdx.y, threadIdx.y, idx, N);
    }
}


int main(int argc,char **argv)
{
    const dim3 blockSize(BLOCK_D1, BLOCK_D2, BLOCK_D3);
    const dim3 gridSize(GRID_D1, GRID_D2, 1);
    int nthreads = BLOCK_D1*BLOCK_D2*BLOCK_D3*GRID_D1*GRID_D2;
    if (nthreads < N){
        printf("\n============ NOT ENOUGH THREADS TO COVER N=%d ===============\n\n",N);
    } else {
        printf("Launching %d threads (N=%d)\n",nthreads,N);
    }
    
    // launch the kernel
    hello<<<gridSize, blockSize>>>();
    
    // Need to flush prints...
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr){
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    } else {
        printf("kernel launch success!\n");
    }
    
    printf("That's all!\n");

    return 0;
}




