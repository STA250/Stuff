
#include <hip/hip_runtime.h>
extern "C"
__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma)
{
       // Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
           /* how big is each block within a grid */
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
            /* get thread within a block */
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

	if(idx < N) {
            float std = (vals[idx] - mu)/sigma;
	    float e = exp( - 0.5 * std * std);
	    vals[idx] = e / ( sigma * sqrt(2 * 3.141592653589793));
	}
}
