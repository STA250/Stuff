#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "utility.h"

extern "C"
{

__global__ void setup_kernel(hiprandState  *state)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    hiprand_init(9131 + idx*17, idx, 0, &state[idx]);
    return;
}

__global__ void runif_kernel(hiprandState *state, float *vals, int n, float lo, float hi)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

    if (idx < n){
        vals[idx] = lo + (hi-lo)*hiprand_uniform(&state[idx]);
    }
    return;
}

__global__ void rexpo_kernel(hiprandState *state, float *vals, int n, float lambda)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

    if (idx < n){
        vals[idx] = -log(hiprand_uniform(&state[idx]))/lambda;
    }
    return;
}

__global__ void rnorm_kernel(hiprandState *state, float *vals, int n, float mu, float sigma)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

	if (idx < n) {
	    vals[idx] = mu + sigma * hiprand_normal(&state[idx]);
	}
    return;
}

__global__ void rpois_kernel(hiprandState *state, int *vals, int n, double lambda)
{
    // TODO: Needs pre-initialization
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

	if (idx < n) {
	    vals[idx] = hiprand_poisson(&state[idx], lambda);
	}
    return;
}

/*
   Few others are available for pseudo-RNGs:

   hiprand_log_normal
   hiprand_uniform_double (double precision uniforms)
   hiprand_normal_double (double precision normals)
   hiprand_log_normal_double (double precision log-normals)
   hiprand_normal2 (pair of float normals)
   hiprand_log_normal2 (pair of float log-normals)
   hiprand_normal2_double (pair of double precision normals)
   hiprand_log_normal2_double (pair of double precision log-normals)
*/

__global__ void rnorm_all_in_one_kernel(float *vals, int n, float mu, float sigma)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

    // Setup the RNG:
    hiprandState rng_state;
    hiprand_init(9131 + idx*17, 0, 0, &rng_state);

	if (idx < n) {
	    vals[idx] = mu + sigma * hiprand_normal(&rng_state);
	}
    return;
}

__global__ void rnorm_improved_kernel(hiprandState *state, int state_len, float *vals, int n, float mu, float sigma)
{
    // 
    // INPUTS
    // ======
    //
    // state        :: RNG states (vector of length state_len)
    //
    // state_len    :: length of rng states (one thread per rng_state)
    //                 if (idx >= state_len){ do nothing... }
    //
    // vals         :: storage for the n samples to be generated
    //
    // n            :: length of vector to sample
    //
    // mu           :: mean
    //
    // sigma        :: standard deviation
    //
    // ===================================================================
    //
    // From these we derive:
    //
    // n_per_thread :: number of rng's to generate per thread
    //                 heurestically, each thread computes
    //                 n_per_thread = n / state_len 
    //                 variates. However, if n_per_thread is
    //                 not an integer, then some do one less,
    //                 some one more.
    //
    // n(idx)       :: number of rng's that thread idx will generate
    //
    // n_lo         :: minimum number of rng's that a (used) thread
    //                 will generate
    //
    // n_hi         :: maximum number of rng's that a (used) thread
    //                 will generate
    //
    // n_extra      :: (= n % state_len) the number of threads that
    //                 will generate n_hi rng's
    //
    // n_regular    :: (= state_len - n_extra) the number of threads that
    //                 will generate n_lo rng's
    // 
    // Note:
    //
    // n = (n_lo * n_regular) + (n_hi * n_extra)
    //
    // e.g.,
    //
    // n = 108, state_len = 40
    // => n_lo = 2, n_hi = 3
    // => n_extra = 28, n_regular = 12
    // => threads  1-28 generate 3 rng's
    // => threads 29-40 generate 2 rngs'
    // 
    // thread idx writes to:
    // 
    // if (idx < n_extra){
    //   // 'extra' thread:
    //   n_hi*idx : ((n_hi*idx) + (n_hi-1))
    // }
    // if (idx >= n_extra){
    //   // 'regular' thread:
    //   (n_hi*n_extra) + (n_lo*(idx-n_extra)):((n_lo*(idx-n_extra)) + (n_lo-1))
    // }
    // 
    // e.g., (cont...)
    //
    // thread  0 writes to: 0,1,2
    // thread  1 writes to: 3,4,5
    // ...
    // thread 27 writes to: 81,82,83
    // thread 28 writes to: 84,85
    // thread 29 writes to: 86,87
    // ...
    // thread 40 writes to: 118,119
    //

    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

	if (idx >= state_len) {
	    return;
	}

    int n_extra, n_lo, n_hi, out_idx_start, n_per_thread;
    compute_tbg_indices(idx,n,state_len,&n_extra,&n_lo,&n_hi,&out_idx_start,&n_per_thread);

    int gen_num;
    for (gen_num=0; gen_num<n_per_thread; gen_num++){
        vals[out_idx_start+gen_num] = mu + sigma * hiprand_normal(&state[idx]);
    }
    return;
}

} // END extern 

